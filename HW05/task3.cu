#include "vscale.cuh"
#include <chrono>
#include <iostream>
#include <cstdlib>
#include <random>
#include <hip/hip_runtime.h>

// Provide some namespace shortcuts
using std::cout;
using std::vector;
using std::chrono::duration;
using std::chrono::high_resolution_clock;

int main(int argc, char *argv[]) {

    if (argc != 2) {
        std::cerr << "Usage: ./task3 n, where n is the length of the array";
        return 1;
    }

    // Parse the arguments
    size_t n = std::atoi(argv[1]);

    // Create two arrays of length n filled with random numbers
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dist(-10.0f, 10.0f);
    std::uniform_real_distribution<float> dist1(0.0f, 1.0f);

    // Allocate memory for host and device arrays
    float *a, *b, *d_a, *d_b;
    hipHostMalloc(&a, n * sizeof(float));  // Pinned host memory
    hipHostMalloc(&b, n * sizeof(float));
    hipMalloc((void**)&d_a, n * sizeof(float));
    hipMalloc((void**)&d_b, n * sizeof(float));

    // Fill host arrays with random values
    for (size_t i = 0; i < n; ++i) {
        a[i] = dist(gen);
        b[i] = dist1(gen);
    }

    // Copy data from host to device
    hipMemcpy(d_a, a, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(float), hipMemcpyHostToDevice);

    // Kernel execution configuration
    int numThreadsPerBlock = 512;
    int numBlocks = (n + numThreadsPerBlock - 1) / numThreadsPerBlock;

    // Set up CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Call the kernel
    vscale<<<numBlocks, numThreadsPerBlock>>>(d_a, d_b, n);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Get the elapsed time in milliseconds
    float ms;
    hipEventElapsedTime(&ms, start, stop);

    // Print the amount of time taken to execute the kernel in milliseconds
    printf("time taken: %f\n", ms);

    // Copy the result from device back to host
    hipMemcpy(b, d_b, n * sizeof(float), hipMemcpyDeviceToHost);

    // Print the first and last elements of the resulting array
    printf("%f\n", b[0]);
    printf("%f\n", b[n - 1]);

    // Free device and host memory
    hipFree(d_a);
    hipFree(d_b);
    hipHostFree(a);
    hipHostFree(b);

    return 0;
}
