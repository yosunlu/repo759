#include "stencil.cuh"
#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>
#include <cstdlib>
#include <random>
#include <hip/hip_runtime.h>

int main(int argc, char *argv[])
{

    // Allocate memory for host and device arrays
    float *h_i, *h_m, *h_o, *d_i, *d_m, *d_o;

    if (hipHostMalloc(&h_i, 10 * sizeof(float)) != hipSuccess)
    {
        std::cerr << "Error allocating pinned memory for array h_i on host\n";
        return 1;
    }

    if (hipHostMalloc(&h_m, 5 * sizeof(float)) != hipSuccess)
    {
        hipHostFree(h_i); // Free previously allocated memory
        std::cerr << "Error allocating pinned memory for array h_m on host\n";
        return 1;
    }

    if (hipHostMalloc(&h_o, 10 * sizeof(float)) != hipSuccess)
    {
        std::cerr << "Error allocating pinned memory for array h_b on host\n";
        hipHostFree(h_i); // Free previously allocated memory
        hipHostFree(h_m);
        return 1;
    }

    if (hipMalloc((void **)&d_i, 10 * sizeof(float)) != hipSuccess)
    {
        std::cerr << "Error allocating memory for array d_i on device\n";
        hipHostFree(h_i);
        hipHostFree(h_m);
        hipHostFree(h_o);
        return 1;
    }

    if (hipMalloc((void **)&d_m, 5 * sizeof(float)) != hipSuccess)
    {
        std::cerr << "Error allocating memory for array d_m on device\n";
        hipHostFree(h_i);
        hipHostFree(h_m);
        hipHostFree(h_o);
        hipFree(d_i);
        return 1;
    }

    if (hipMalloc((void **)&d_o, 10 * sizeof(float)) != hipSuccess)
    {
        std::cerr << "Error allocating memory for array d_o on device\n";
        hipHostFree(h_i);
        hipHostFree(h_m);
        hipHostFree(h_o);
        hipFree(d_i);
        hipFree(d_m);
        return 1;
    }

    // Fill host image with  values
    for (size_t i = 0; i < 10; ++i)
    {
        h_i[i] = i;
    }

    // Fill host mask with  values
    for (int i = 0; i < 5; ++i)
    {
        h_m[i] = -1 * i; 
        // std::cout << h_m[i] << std::endl; 
    }

    // Copy data from host to device
    hipMemcpy(d_i, h_i, 10 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_m, h_m, 5 * sizeof(float), hipMemcpyHostToDevice);

    // call the stencil function
    stencil(d_i, d_m, d_o, 10, 2, 5);

    // Copy data from device back to host
    hipMemcpy(h_o, d_o, 5 * sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 0; i < 10; ++i)
    {
        std::cout << h_o[i] << std::endl;
    }

    // Free device and host memory
    hipFree(d_i);
    hipFree(d_m);
    hipFree(d_o);
    hipHostFree(h_i);
    hipHostFree(h_m);
    hipHostFree(h_o);


}