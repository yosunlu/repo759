#include "stencil.cuh"
#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>
#include <cstdlib>
#include <random>
#include <hip/hip_runtime.h>

int main(int argc, char *argv[])
{

    if (argc != 4)
    {
        std::cerr << "Usage: ./task2 n R threads_per_block, where 2 * R +1 is the length of the mask, and n is the length of the array";
        return 1;
    }

    // Parse the arguments
    size_t n = std::atoi(argv[1]); 
    unsigned int R = std::atoi(argv[2]);;
    size_t threads_per_block = std::atoi(argv[3]);

    // Allocate memory for host and device arrays
    float *h_i, *h_m, *h_o, *d_i, *d_m, *d_o;

    if (hipHostMalloc(&h_i, n * sizeof(float)) != hipSuccess)
    {
        std::cerr << "Error allocating pinned memory for array h_i on host\n";
        return 1;
    }

    if (hipHostMalloc(&h_m, (2 * R + 1) * sizeof(float)) != hipSuccess)
    {
        hipHostFree(h_i); // Free previously allocated memory
        std::cerr << "Error allocating pinned memory for array h_m on host\n";
        return 1;
    }

    if (hipHostMalloc(&h_o, n * sizeof(float)) != hipSuccess)
    {
        std::cerr << "Error allocating pinned memory for array h_b on host\n";
        hipHostFree(h_i); // Free previously allocated memory
        hipHostFree(h_m);
        return 1;
    }

    if (hipMalloc((void **)&d_i, n * sizeof(float)) != hipSuccess)
    {
        std::cerr << "Error allocating memory for array d_i on device\n";
        hipHostFree(h_i);
        hipHostFree(h_m);
        hipHostFree(h_o);
        return 1;
    }

    if (hipMalloc((void **)&d_m, (2 * R + 1) * sizeof(float)) != hipSuccess)
    {
        std::cerr << "Error allocating memory for array d_m on device\n";
        hipHostFree(h_i);
        hipHostFree(h_m);
        hipHostFree(h_o);
        hipFree(d_i);
        return 1;
    }

    if (hipMalloc((void **)&d_o, n * sizeof(float)) != hipSuccess)
    {
        std::cerr << "Error allocating memory for array d_o on device\n";
        hipHostFree(h_i);
        hipHostFree(h_m);
        hipHostFree(h_o);
        hipFree(d_i);
        hipFree(d_m);
        return 1;
    }

    // Fill the image and mask array with random numbers in the range [-1, 1].
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dist(-1.0f, 1.0f);

    // Fill host image with  values
    for (size_t i = 0; i < n; ++i)
    {
        h_i[i] = dist(gen);
    }

    // Fill host mask with  values
    for (int i = 0; i < (int)R; ++i)
    {
        h_m[i] = dist(gen); 
    }

    // Copy data from host to device
    hipMemcpy(d_i, h_i, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_m, h_m, (2 * R + 1) * sizeof(float), hipMemcpyHostToDevice);


    // Set up CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // call the stencil function
    stencil(d_i, d_m, d_o, n, R, threads_per_block);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    // Get the elapsed time in milliseconds
    float ms;
    hipEventElapsedTime(&ms, start, stop);

    // Copy data from device back to host
    hipMemcpy(h_o, d_o, n * sizeof(float), hipMemcpyDeviceToHost);
 
    // Print the last element of the output matrix.
    printf("%f\n", h_o[n - 1]);

    // Print the amount of time taken to execute the kernel in milliseconds
    printf("time taken: %f\n", ms);

    // Free device and host memory
    hipFree(d_i);
    hipFree(d_m);
    hipFree(d_o);
    hipHostFree(h_i);
    hipHostFree(h_m);
    hipHostFree(h_o);

}