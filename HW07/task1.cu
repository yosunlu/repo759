#include "matmul.cuh"
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <hip/hip_runtime.h>

template <typename T>
void initialize_matrix(T *matrix, size_t size, T min_val, T max_val) {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<> dist(min_val, max_val);

    for (size_t i = 0; i < size; ++i) {
        matrix[i] = static_cast<T>(dist(gen));
    }
}

template <typename T>
void run_matmul(void (*matmul_func)(const T *, const T *, T *, unsigned int, unsigned int),
                const char *func_name, unsigned int n, unsigned int block_dim) {
    size_t size = n * n;
    size_t bytes = size * sizeof(T);

    // Allocate host memory
    T *h_a, *h_b, *h_c;
    hipHostMalloc(&h_a, bytes);
    hipHostMalloc(&h_b, bytes);
    hipHostMalloc(&h_c, bytes);

    // Initialize matrices with random values
    initialize_matrix(h_a, size, static_cast<T>(-1), static_cast<T>(1));
    initialize_matrix(h_b, size, static_cast<T>(-1), static_cast<T>(1));

    // Allocate device memory
    T *d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, bytes);
    hipMalloc((void **)&d_b, bytes);
    hipMalloc((void **)&d_c, bytes);

    // Copy matrices to device
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

    // Set up CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Call matmul function and measure time
    hipEventRecord(start);
    matmul_func(d_a, d_b, d_c, n, block_dim);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Get elapsed time
    float ms;
    hipEventElapsedTime(&ms, start, stop);

    // Copy result back to host
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    // Print results
    std::cout << "Results for " << func_name << ":\n";
    std::cout << "First element of C: " << h_c[0] << "\n";
    std::cout << "Last element of C: " << h_c[size - 1] << "\n";
    std::cout << "Time taken: " << ms << " ms\n";

    // Free memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);

    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

int main(int argc, char *argv[]) {
    if (argc != 3) {
        std::cerr << "Usage: ./task1 n block_dim\n";
        return 1;
    }

    // Parse arguments
    unsigned int n = std::atoi(argv[1]);
    unsigned int block_dim = std::atoi(argv[2]);

    // Run tests for matmul_1, matmul_2, and matmul_3
    run_matmul<int>(matmul_1, "matmul_1 (int)", n, block_dim);
    run_matmul<float>(matmul_2, "matmul_2 (float)", n, block_dim);
    run_matmul<double>(matmul_3, "matmul_3 (double)", n, block_dim);

    return 0;
}
