#include "matmul.cuh"
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <hip/hip_runtime.h>

// int run_matmul_int(unsigned int n, unsigned int size, unsigned int block_dim)
// {
//     // generate random variables
//     std::random_device rd;
//     std::mt19937 gen(rd());
//     std::uniform_int_distribution<int> dist(-10, 10);

//     // Allocate memory for host and device arrays
//     int *h_a, *h_b, *h_c, *d_a, *d_b, *d_c;
//     if (hipHostMalloc(&h_a, size * sizeof(int)) != hipSuccess)
//     {
//         std::cerr << "Error allocating pinned memory for array h_a on host\n";
//         return 1;
//     }
//     if (hipHostMalloc(&h_b, size * sizeof(int)) != hipSuccess)
//     {
//         std::cerr << "Error allocating pinned memory for array h_b on host\n";
//         hipHostFree(h_a); // Free previously allocated memory
//         return 1;
//     }
//     if (hipHostMalloc(&h_c, size * sizeof(int)) != hipSuccess)
//     {
//         std::cerr << "Error allocating pinned memory for array h_b on host\n";
//         hipHostFree(h_a); // Free previously allocated memory
//         hipHostFree(h_b);
//         return 1;
//     }
//     if (hipMalloc((void **)&d_a, size * sizeof(int)) != hipSuccess)
//     {
//         std::cerr << "Error allocating memory for array d_a on device\n";
//         hipHostFree(h_a);
//         hipHostFree(h_b);
//         hipHostFree(h_c);
//         return 1;
//     }
//     if (hipMalloc((void **)&d_b, size * sizeof(int)) != hipSuccess)
//     {
//         std::cerr << "Error allocating memory for array d_b on device\n";
//         hipHostFree(h_a);
//         hipHostFree(h_b);
//         hipHostFree(h_c);
//         hipFree(d_a);
//         return 1;
//     }
//     if (hipMalloc((void **)&d_c, size * sizeof(int)) != hipSuccess)
//     {
//         std::cerr << "Error allocating memory for array d_c on device\n";
//         hipHostFree(h_a);
//         hipHostFree(h_b);
//         hipHostFree(h_c);
//         hipFree(d_a);
//         hipFree(d_b);
//         return 1;
//     }

//     // Fill host arrays with random values
//     for (size_t i = 0; i < size; ++i)
//     {
//         h_a[i] = dist(gen);
//         h_b[i] = dist(gen);
//     }

//     // Copy data from host to device
//     hipMemcpy(d_a, h_a, size * sizeof(int), hipMemcpyHostToDevice);
//     hipMemcpy(d_b, h_b, size * sizeof(int), hipMemcpyHostToDevice);

//     // Set up CUDA events for timing
//     hipEvent_t start, stop;
//     hipEventCreate(&start);
//     hipEventCreate(&stop);
//     hipEventRecord(start);

//     // Run tests for matmul_1, matmul_2, and matmul_3
//     matmul_1(d_a, d_b, d_c, n, block_dim);

//     hipEventRecord(stop);
//     hipEventSynchronize(stop);

//     // Get the elapsed time in milliseconds
//     float ms;
//     hipEventElapsedTime(&ms, start, stop);

//     // Copy the result from device back to host
//     hipMemcpy(h_c, d_c, size * sizeof(int), hipMemcpyDeviceToHost);

//     // Print the last element of the resulting matrix.
//     printf("first element: %d\n", h_c[0]);

//     // Print the last element of the resulting matrix.
//     printf("last element: %d\n", h_c[size - 1]);

//     // Print the amount of time taken to execute the kernel in milliseconds
//     printf("time taken: %f\n", ms);

//     hipFree(d_a);
//     hipFree(d_b);
//     hipFree(d_c);
//     hipHostFree(h_a);
//     hipHostFree(h_b);
//     hipHostFree(h_c);

//     return 0;
// }

// int run_matmul_float(unsigned int n, unsigned int size, unsigned int block_dim)
// {
//     // Generate random variables
//     std::random_device rd;
//     std::mt19937 gen(rd());
//     std::uniform_real_distribution<float> dist(-10.0f, 10.0f);

//     // Allocate memory for host and device arrays
//     float *h_a, *h_b, *h_c, *d_a, *d_b, *d_c;
//     if (hipHostMalloc(&h_a, size * sizeof(float)) != hipSuccess)
//     {
//         std::cerr << "Error allocating pinned memory for array h_a on host\n";
//         return 1;
//     }
//     if (hipHostMalloc(&h_b, size * sizeof(float)) != hipSuccess)
//     {
//         std::cerr << "Error allocating pinned memory for array h_b on host\n";
//         hipHostFree(h_a);
//         return 1;
//     }
//     if (hipHostMalloc(&h_c, size * sizeof(float)) != hipSuccess)
//     {
//         std::cerr << "Error allocating pinned memory for array h_c on host\n";
//         hipHostFree(h_a);
//         hipHostFree(h_b);
//         return 1;
//     }
//     if (hipMalloc((void **)&d_a, size * sizeof(float)) != hipSuccess)
//     {
//         std::cerr << "Error allocating memory for array d_a on device\n";
//         hipHostFree(h_a);
//         hipHostFree(h_b);
//         hipHostFree(h_c);
//         return 1;
//     }
//     if (hipMalloc((void **)&d_b, size * sizeof(float)) != hipSuccess)
//     {
//         std::cerr << "Error allocating memory for array d_b on device\n";
//         hipHostFree(h_a);
//         hipHostFree(h_b);
//         hipHostFree(h_c);
//         hipFree(d_a);
//         return 1;
//     }
//     if (hipMalloc((void **)&d_c, size * sizeof(float)) != hipSuccess)
//     {
//         std::cerr << "Error allocating memory for array d_c on device\n";
//         hipHostFree(h_a);
//         hipHostFree(h_b);
//         hipHostFree(h_c);
//         hipFree(d_a);
//         hipFree(d_b);
//         return 1;
//     }

//     // Fill host arrays with random values
//     for (size_t i = 0; i < size; ++i)
//     {
//         h_a[i] = dist(gen);
//         h_b[i] = dist(gen);
//     }

//     // Copy data from host to device
//     hipMemcpy(d_a, h_a, size * sizeof(float), hipMemcpyHostToDevice);
//     hipMemcpy(d_b, h_b, size * sizeof(float), hipMemcpyHostToDevice);

//     // Set up CUDA events for timing
//     hipEvent_t start, stop;
//     hipEventCreate(&start);
//     hipEventCreate(&stop);
//     hipEventRecord(start);

//     // Run the kernel
//     matmul_2(d_a, d_b, d_c, n, block_dim);

//     hipEventRecord(stop);
//     hipEventSynchronize(stop);

//     // Get the elapsed time in milliseconds
//     float ms;
//     hipEventElapsedTime(&ms, start, stop);

//     // Copy the result from device back to host
//     hipMemcpy(h_c, d_c, size * sizeof(float), hipMemcpyDeviceToHost);

//     // Print the first and last elements of the resulting matrix
//     printf("First element: %f\n", h_a[0]);
//     printf("Last element: %f\n", h_a[size - 1]);

//     // Print the amount of time taken to execute the kernel
//     printf("Time taken: %f ms\n", ms);

//     // Cleanup
//     hipFree(d_a);
//     hipFree(d_b);
//     hipFree(d_c);
//     hipHostFree(h_a);
//     hipHostFree(h_b);
//     hipHostFree(h_c);

//     return 0;
// }

int run_matmul_double(unsigned int n, unsigned int size, unsigned int block_dim)
{
    // Generate random variables
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<double> dist(-10.0, 10.0);

    // Allocate memory for host and device arrays
    double *h_a, *h_b, *h_c, *d_a, *d_b, *d_c;
    if (hipHostMalloc(&h_a, size * sizeof(double)) != hipSuccess)
    {
        std::cerr << "Error allocating pinned memory for array h_a on host\n";
        return 1;
    }
    if (hipHostMalloc(&h_b, size * sizeof(double)) != hipSuccess)
    {
        std::cerr << "Error allocating pinned memory for array h_b on host\n";
        hipHostFree(h_a);
        return 1;
    }
    if (hipHostMalloc(&h_c, size * sizeof(double)) != hipSuccess)
    {
        std::cerr << "Error allocating pinned memory for array h_c on host\n";
        hipHostFree(h_a);
        hipHostFree(h_b);
        return 1;
    }
    if (hipMalloc((void **)&d_a, size * sizeof(double)) != hipSuccess)
    {
        std::cerr << "Error allocating memory for array d_a on device\n";
        hipHostFree(h_a);
        hipHostFree(h_b);
        hipHostFree(h_c);
        return 1;
    }
    if (hipMalloc((void **)&d_b, size * sizeof(double)) != hipSuccess)
    {
        std::cerr << "Error allocating memory for array d_b on device\n";
        hipHostFree(h_a);
        hipHostFree(h_b);
        hipHostFree(h_c);
        hipFree(d_a);
        return 1;
    }
    if (hipMalloc((void **)&d_c, size * sizeof(double)) != hipSuccess)
    {
        std::cerr << "Error allocating memory for array d_c on device\n";
        hipHostFree(h_a);
        hipHostFree(h_b);
        hipHostFree(h_c);
        hipFree(d_a);
        hipFree(d_b);
        return 1;
    }

    // Fill host arrays with random values
    for (size_t i = 0; i < size; ++i)
    {
        h_a[i] = dist(gen);
        h_b[i] = dist(gen);
    }

    // Copy data from host to device
    hipMemcpy(d_a, h_a, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size * sizeof(double), hipMemcpyHostToDevice);

    // Set up CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Run the kernel
    matmul_3(d_a, d_b, d_c, n, block_dim);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Get the elapsed time in milliseconds
    float ms;
    hipEventElapsedTime(&ms, start, stop);

    // Copy the result from device back to host
    hipMemcpy(h_c, d_c, size * sizeof(double), hipMemcpyDeviceToHost);

    // Print the first and last elements of the resulting matrix
    printf("First element: %f\n", h_a[0]);
    printf("Last element: %f\n", h_a[size - 1]);

    // Print the amount of time taken to execute the kernel
    printf("Time taken: %f ms\n", ms);

    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);

    return 0;
}

int main(int argc, char *argv[])
{
    if (argc != 3)
    {
        std::cerr << "Usage: ./task1 n block_dim\n";
        return 1;
    }

    // Parse arguments
    unsigned int n = std::atoi(argv[1]);
    unsigned int size = n * n;
    unsigned int block_dim = std::atoi(argv[2]);

    // run_matmul_int(n, size, block_dim);
    // run_matmul_float(n, size, block_dim);
    run_matmul_double(n, size, block_dim);

    return 0;
}
