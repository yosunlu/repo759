#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <hip/hip_runtime.h>
#include "reduce.cuh"

int main(int argc, char *argv[])
{
    if (argc != 3)
    {
        std::cerr << "Usage: ./task2 N threads_per_block\n";
        return 1;
    }

    // Parse the arguments
    size_t N = std::atoi(argv[1]);
    size_t threads_per_block = std::atoi(argv[2]);

    // Allocate memory for host and device arrays
    float *h_input, *h_output, *d_input, *d_output;

    if (hipHostMalloc(&h_input, N * sizeof(float)) != hipSuccess)
    {
        std::cerr << "Error allocating pinned memory for input array on host\n";
        return 1;
    }

    size_t num_blocks = (N + threads_per_block * 2 - 1) / (threads_per_block * 2);

    if (hipHostMalloc(&h_output, num_blocks * sizeof(float)) != hipSuccess)
    {
        std::cerr << "Error allocating pinned memory for output array on host\n";
        hipHostFree(h_input);
        return 1;
    }

    if (hipMalloc((void **)&d_input, N * sizeof(float)) != hipSuccess)
    {
        std::cerr << "Error allocating memory for input array on device\n";
        hipHostFree(h_input);
        hipHostFree(h_output);
        return 1;
    }

    if (hipMalloc((void **)&d_output, num_blocks * sizeof(float)) != hipSuccess)
    {
        std::cerr << "Error allocating memory for output array on device\n";
        hipHostFree(h_input);
        hipHostFree(h_output);
        hipFree(d_input);
        return 1;
    }

    // Fill the input array with random numbers in the range [-1, 1].
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dist(-1.0f, 1.0f);

    for (size_t i = 0; i < N; ++i)
    {
        h_input[i] = dist(gen);
    }

    // Copy data from host to device
    hipMemcpy(d_input, h_input, N * sizeof(float), hipMemcpyHostToDevice);

    // Set up CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Call the reduce function
    reduce(&d_input, &d_output, N, threads_per_block);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Get the elapsed time in milliseconds
    float ms;
    hipEventElapsedTime(&ms, start, stop);

    // Copy the result from device back to host
    hipMemcpy(h_output, d_output, num_blocks * sizeof(float), hipMemcpyDeviceToHost);

    // Print the resulting sum
    printf("Resulting sum: %f\n", h_output[0]);

    // Print the time taken
    printf("Time taken: %f ms\n", ms);

    // Free device and host memory
    hipFree(d_input);
    hipFree(d_output);
    hipHostFree(h_input);
    hipHostFree(h_output);

    return 0;
}